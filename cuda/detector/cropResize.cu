#include "hip/hip_runtime.h"
/*
Kernel to crop and resize boxes from an image
*/
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>

#include "common.h"

using namespace std;

// blockSize will be {1024, 1, 1}
// gridSize...
__global__ void cropResizeKernel(
    const float *image, int imageWidth, int imageHeight, int depth,
    const float *boxes, int boxesSize, int cropWidth, int cropHeight,
    float *croppedBoxes,
    int croppedBoxesSize // == boxesSize * cropWidth * cropHeight * depth
) {
  // Assume that the depth is 3 for now

  const float extrapolationValue = 0.0f;
  const int batch = 1;

  // Each thread will loop and write to certain Idx in croppedBoxes
  for (int outIdx = threadIdx.x + blockIdx.x * blockDim.x;
       outIdx < croppedBoxesSize; outIdx += blockDim.x * gridDim.x) {
    int idx = outIdx;
    const int x = idx % cropWidth;
    idx /= cropWidth;
    const int y = idx % cropHeight;
    idx /= cropHeight;
    const int depthIdx = idx % depth;
    const int boxIdx = idx / depth;

    const float y1 = boxes[boxIdx * 4];
    const float x1 = boxes[boxIdx * 4 + 1];
    const float y2 = boxes[boxIdx * 4 + 2];
    const float x2 = boxes[boxIdx * 4 + 3];

    const int batchIdx = boxIdx / boxesSize;
    if (batchIdx < 0 || batchIdx >= batch) {
      printf("Unexpected batchIdx: %d\n", batchIdx);
      continue;
    }

    const float heightScale =
        (cropHeight > 1) ? (y2 - y1) * (imageHeight - 1) / (cropHeight - 1) : 0;
    const float widthScale =
        (cropWidth > 1) ? (x2 - x1) * (imageWidth - 1) / (cropWidth - 1) : 0;

    const float inY = (cropHeight > 1)
                          ? y1 * (imageHeight - 1) + y * heightScale
                          : 0.5 * (y1 + y2) * (imageHeight - 1);
    if (inY < 0 || inY > imageHeight - 1) {
      croppedBoxes[outIdx] = extrapolationValue;
      continue;
    }

    const float inX = (cropWidth > 1) ? x1 * (imageWidth - 1) + x * widthScale
                                      : 0.5 * (x1 + x2) * (imageWidth - 1);
    if (inX < 0 || inX > imageWidth - 1) {
      croppedBoxes[outIdx] = extrapolationValue;
      continue;
    }

    const int topYIndex = floorf(inY);
    const int bottomYIndex = ceilf(inY);
    const float yLerp = inY - topYIndex;
    const int leftXIndex = floorf(inX);
    const int rightXIndex = ceilf(inX);
    const float xLerp = inX - leftXIndex;

    const float topLeft(static_cast<float>(
        image[((batchIdx * depth + depthIdx) * imageHeight + topYIndex) *
                  imageWidth +
              leftXIndex]));
    const float topRight(static_cast<float>(
        image[((batchIdx * depth + depthIdx) * imageHeight + topYIndex) *
                  imageWidth +
              rightXIndex]));
    const float bottomLeft(static_cast<float>(
        image[((batchIdx * depth + depthIdx) * imageHeight + bottomYIndex) *
                  imageWidth +
              leftXIndex]));
    const float bottomRight(static_cast<float>(
        image[((batchIdx * depth + depthIdx) * imageHeight + bottomYIndex) *
                  imageWidth +
              rightXIndex]));
    const float top = topLeft + (topRight - topLeft) * xLerp;
    const float bottom = bottomLeft + (bottomRight - bottomLeft) * xLerp;
    croppedBoxes[out_idx] = top + (bottom - top) * yLerp;
  }
}

vector<float> runCropResize(
  const vector<float> &image, int imageWidth, int imageHeight, int depth,
  const vecotr<float> &boxes, int boxesSize, int cropWidth, int cropHeight
) {
  int croppedBoxesSize = boxesSize * cropWidth * cropHeight * depth;
  vector<float> croppedBoxes(croppedBoxesSize);

  float *dImage;
  float *dBoxes;
  float *dCroppedBoxes;

  CUDACHECK(hipMalloc((void **)&dImage, sizeof(float) * image.size()));
  CUDACHECK(hipMalloc((void **)&dBoxes, sizeof(float) * boxes.size()));
  CUDACHECK(hipMalloc((void **)&dCroppedBoxes, sizeof(float) * croppedBoxes.size()));

  CUDACHECK(hipMemcpy((void *)dImage, (void *)image.data()),
            sizeof(float) * image.size(), hipMemcpyHostToDevice);

  const int blockSize = 1024;
  const int gridSize = (croppedBoxesSize + blockSize - 1) / blockSize;

  cropResizeKernel<<<grid, block>>>(
    dImage, imageWidth, imageHeight, depth,
    dBoxes, boxesSize, cropWidth, cropHeight,
    dCroppedBoxes, croppedBoxesSize
  );

  CUDACHECK(hipMemcpy((void *)croppedBoxes.data(), (void *)dCroppedBoxes,
    sizeof(float) * croppedBoxes.size(), hipMemcpyDeviceToHost));

  CUDACHECK(hipFree((void *)dImage));
  CUDACHECK(hipFree((void *)dBoxes));
  CUDACHECK(hipFree((void *)dCroppedBoxes));

  return croppedBoxes;
}


int main(int argc, char **argv) {
  int imageWidth = 2;
  int imageHeight = 2;
  int depth = 3;
  vector<float> image {
    1.0, 1.0, 1.0,
    2.0, 2.0, 2.0,
    3.0, 3.0, 3.0,
    4.0, 4.0, 4.0
  };

  vector<float> boxes = { 0.3, 0.3, 0.7, 0.7};
  int boxesSize = 1;
  int cropHeight = 50;
  int cropWidth = 50;

  auto croppedBoxes = runCropResize(
    image, imageWidth, imageHeight, depth,
    boxes, boxesSize, cropHeight, cropWidth
  );

  cout << croppedBoxesSize.size() << endl;

  return 0;
}
