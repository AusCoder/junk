#include "hip/hip_runtime.h"
/*
Kernel to crop and resize boxes from an image
*/
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>

#include "common.h"

using namespace std;

// blockSize will be {1024, 1, 1}
// gridSize...
__global__ void cropResizeKernel(
    const float *image, int imageWidth, int imageHeight, int depth,
    const float *boxes, int boxesSize, int cropWidth, int cropHeight,
    float *croppedBoxes,
    int croppedBoxesSize // == boxesSize * cropWidth * cropHeight * depth
) {
  // Assume that the depth is 3 for now

  const float extrapolationValue = 0.0f;
  const int batch = 1;

  // Each thread will loop and write to certain Idx in croppedBoxes
  for (int outIdx = threadIdx.x + blockIdx.x * blockDim.x;
       outIdx < croppedBoxesSize; outIdx += blockDim.x * gridDim.x) {
    int idx = outIdx;
    const int x = idx % cropWidth;
    idx /= cropWidth;
    const int y = idx % cropHeight;
    idx /= cropHeight;
    const int depthIdx = idx % depth;
    const int boxIdx = idx / depth;

    const float y1 = boxes[boxIdx * 4];
    const float x1 = boxes[boxIdx * 4 + 1];
    const float y2 = boxes[boxIdx * 4 + 2];
    const float x2 = boxes[boxIdx * 4 + 3];

    const int batchIdx = boxIdx / boxesSize;
    if (batchIdx < 0 || batchIdx >= batch) {
      printf("Unexpected batchIdx: %d\n", batchIdx);
      continue;
    }

    const float heightScale =
        (cropHeight > 1) ? (y2 - y1) * (imageHeight - 1) / (cropHeight - 1) : 0;
    const float widthScale =
        (cropWidth > 1) ? (x2 - x1) * (imageWidth - 1) / (cropWidth - 1) : 0;

    const float inY = (cropHeight > 1)
                          ? y1 * (imageHeight - 1) + y * heightScale
                          : 0.5 * (y1 + y2) * (imageHeight - 1);
    if (inY < 0 || inY > imageHeight - 1) {
      croppedBoxes[outIdx] = extrapolationValue;
      continue;
    }

    const float inX = (cropWidth > 1) ? x1 * (imageWidth - 1) + x * widthScale
                                      : 0.5 * (x1 + x2) * (imageWidth - 1);
    if (inX < 0 || inX > imageWidth - 1) {
      croppedBoxes[outIdx] = extrapolationValue;
      continue;
    }

    const int topYIndex = floorf(inY);
    const int bottomYIndex = ceilf(inY);
    const float yLerp = inY - topYIndex;
    const int leftXIndex = floorf(inX);
    const int rightXIndex = ceilf(inX);
    const float xLerp = inX - leftXIndex;

    const float topLeft(static_cast<float>(
        image[((batchIdx * depth + depthIdx) * imageHeight + topYIndex) *
                  imageWidth +
              leftXIndex]));
    const float topRight(static_cast<float>(
        image[((batchIdx * depth + depthIdx) * imageHeight + topYIndex) *
                  imageWidth +
              rightXIndex]));
    const float bottomLeft(static_cast<float>(
        image[((batchIdx * depth + depthIdx) * imageHeight + bottomYIndex) *
                  imageWidth +
              leftXIndex]));
    const float bottomRight(static_cast<float>(
        image[((batchIdx * depth + depthIdx) * imageHeight + bottomYIndex) *
                  imageWidth +
              rightXIndex]));
    const float top = topLeft + (topRight - topLeft) * xLerp;
    const float bottom = bottomLeft + (bottomRight - bottomLeft) * xLerp;
    croppedBoxes[outIdx] = top + (bottom - top) * yLerp;
  }
}

vector<float> runCropResize(const vector<float> &image, int imageWidth,
                            int imageHeight, int depth,
                            const vector<float> &boxes, int boxesSize,
                            int cropWidth, int cropHeight) {
  int croppedBoxesSize = boxesSize * cropWidth * cropHeight * depth;
  vector<float> croppedBoxes(croppedBoxesSize);

  float *dImage;
  float *dBoxes;
  float *dCroppedBoxes;

  CUDACHECK(hipMalloc((void **)&dImage, sizeof(float) * image.size()));
  CUDACHECK(hipMalloc((void **)&dBoxes, sizeof(float) * boxes.size()));
  CUDACHECK(
      hipMalloc((void **)&dCroppedBoxes, sizeof(float) * croppedBoxes.size()));

  CUDACHECK(hipMemcpy((void *)dImage, (void *)image.data(),
                       sizeof(float) * image.size(), hipMemcpyHostToDevice));

  const int block = 1024;
  const int grid = (croppedBoxesSize + block - 1) / block;

  cropResizeKernel<<<grid, block>>>(dImage, imageWidth, imageHeight, depth,
                                    dBoxes, boxesSize, cropWidth, cropHeight,
                                    dCroppedBoxes, croppedBoxesSize);

  CUDACHECK(hipMemcpy((void *)croppedBoxes.data(), (void *)dCroppedBoxes,
                       sizeof(float) * croppedBoxes.size(),
                       hipMemcpyDeviceToHost));

  CUDACHECK(hipFree((void *)dImage));
  CUDACHECK(hipFree((void *)dBoxes));
  CUDACHECK(hipFree((void *)dCroppedBoxes));

  return croppedBoxes;
}

int main(int argc, char **argv) {
  // int imageWidth = 2;
  // int imageHeight = 2;
  // int depth = 3;
  // vector<float> image{1.0, 1.0, 1.0, 2.0, 2.0, 2.0,
  //                     3.0, 3.0, 3.0, 4.0, 4.0, 4.0};

  std::string imagePath =
      "/home/seb/code/ii/ml-source/data/sample_images/execs.jpg";
  cv::Mat image = cv::imread(imagePath, cv::IMREAD_COLOR);
  // image.convertTo(image, cv::CV_32FC3);

  cv::Vec3f point = image.at<cv::Vec3f>(0, 0);
  cout << point[0] << endl;

  // vector<float> boxes = {0.3, 0.3, 0.7, 0.7};
  // int boxesSize = 1;
  // int cropHeight = 9;
  // int cropWidth = 9;

  // auto croppedBoxes = runCropResize(image, imageWidth, imageHeight, depth,
  //                                   boxes, boxesSize, cropHeight, cropWidth);

  // for (auto &val : croppedBoxes) {
  //   cout << val << ", ";
  // }
  // cout << endl;

  return 0;
}
