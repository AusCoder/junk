#include "hip/hip_runtime.h"
/*
Program to attempt mtcnn box generation in cuda.
*/
#include <iostream>
#include <iterator>
#include <sstream>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>

#include "cnpy.h"
#include "common.h"

using namespace std;

/*
  Here every thread is looping the whole prob array, I don't
  know of a way around this from a fundamental level.

  I can think of us splitting the prob grid up a lot then looping in
  each block.

  The issue with doing it this way is that we can't hit that much of the
  prob array. Here if we run this kernel with a block size of {256, 1, 1}
  then we will hit just the first 256 prob entries, it might be much bigger
  than this. To get around this, I have seen other nms code create 2 nested
  loops. With the outer one striding over a small number of elements.

  See nmsLayer.cu in the TensorRT kernels.
*/
__global__ void generateBoxesKernelSimple(Prob *prob, int probWidth,
                                          int probHeight, int *outIndices,
                                          int maxOutIndices) {
  // worry about the blockIdx offset later

  // NB: Here we need the blockDim.y to be less than probWidth,
  // Otherwise we get the same threadIdx from 2 different index combinations
  int thisIdx = threadIdx.y * probWidth + threadIdx.x;
  int probSize = probWidth * probHeight;
  __shared__ int outIdx;
  if (threadIdx.z == 0 && threadIdx.y == 0 && threadIdx.x == 0) {
    outIdx = 0;
  }

  __syncthreads();

  for (int i = 0; i < probSize; i++) {
    if (thisIdx == i) {
      Prob p = prob[thisIdx];
      if (p.y > 0.5) {
        outIndices[outIdx] = thisIdx;
        printf("Gpu. thisIdx: %d. outIdx: %d\n", thisIdx, outIdx);
        outIdx++;
      }
    }
    __syncthreads();
    if (outIdx == maxOutIndices) {
      return;
    }
  }
}

// DIM is going to be the blockSize, ie blockDim.x
// I have seen it templated for loop unrolling?
template <int TSIZE, int DIM>
__global__ void generateBoxesKernel(Prob *prob, int probWidth, int probHeight,
                                    int *outIndices, int maxOutIndices) {
  // This is for a single element, ie a batch size of 1
  // I have seen nms code that uses one block per batch item
  // See nmsLayer.cu from TensorRT kernels

  Prob thisThreadProbs[TSIZE];

  __shared__ int outIdx;
  if (threadIdx.z == 0 && threadIdx.y == 0 && threadIdx.x == 0) {
    outIdx = 0;
  }

  int probSize = probWidth * probHeight;

  for (int i = 0; i < TSIZE; i++) {
    if (i * DIM + threadIdx.x < probSize) {
      thisThreadProbs[i] = prob[i * DIM + threadIdx.x];
    }
  }

  for (int i = 0; i < TSIZE; i++) {

    for (int j = 0; j < DIM; j++) {

      int offset = i * DIM;
      int index = offset + j;
      if (index >= probSize) {
        break;
      }

      __syncthreads();

      if (threadIdx.x == j) {
        Prob p = thisThreadProbs[i];
        if (p.y > 0.95) {
          outIndices[outIdx] = index;
          printf("Gpu. index: %d. outIdx: %d\n", index, outIdx);
          outIdx++;
        }
      }

      __syncthreads();

      if (outIdx == maxOutIndices) {
        return;
      }
    }
  }
}

vector<int> getIndicesAboveThreshold(const vector<Prob> &prob, int width,
                                     int height, int maxOutIndices) {
  vector<int> outIndices(maxOutIndices);
  Prob *dProb;
  int *dOutIndices;

  CUDACHECK(hipMalloc((void **)&dProb, sizeof(Prob) * prob.size()));
  CUDACHECK(hipMalloc((void **)&dOutIndices, sizeof(int) * outIndices.size()));

  CUDACHECK(hipMemcpy((void *)dProb, (void *)prob.data(),
                       sizeof(Prob) * prob.size(), hipMemcpyHostToDevice));

  int grid = 1;
  const int block = 1024;
  const int tsize = 60;

  generateBoxesKernel<tsize, block>
      <<<grid, block>>>(dProb, width, height, dOutIndices, outIndices.size());

  CUDACHECK(hipMemcpy((void *)outIndices.data(), (void *)dOutIndices,
                       sizeof(int) * outIndices.size(),
                       hipMemcpyDeviceToHost));

  CUDACHECK(hipFree((void *)dProb));
  CUDACHECK(hipFree((void *)dOutIndices));

  return outIndices;
}

int main(int argc, char **argv) {
  // int width = 2;
  // int height = 2;
  // vector<Prob> prob{{0.1, 0.9}, {0.8, 0.2}, {0.4, 0.6}, {0.3, 0.7}};
  // int maxOutIndices = 3;

  vector<Prob> prob;
  int maxOutIndices = 500;

  char arrayFilename[] =
      "./mtcnn-output-arrays/stage-one/prob-0.npy";
  cnpy::NpyArray arr = cnpy::npy_load(arrayFilename);
  vector<float> items = arr.as_vec<float>();
  int height = arr.shape.at(1);
  int width = arr.shape.at(2);

  // auto it = items.begin();
  assert(items.size() % 2 == 0);
  for (auto it = items.begin(); it != items.end();) {
    prob.emplace_back(*it, *(it + 1));
    advance(it, 2);
  }

  auto outIndices =
      getIndicesAboveThreshold(prob, width, height, maxOutIndices);

  for (auto &i : outIndices) {
    cout << i << endl;
  }
}
